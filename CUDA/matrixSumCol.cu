
#include <hip/hip_runtime.h>
#include <iostream>

void check(const char *file, const int line, hipError_t err) {
	if (err != hipSuccess) {
		std::cerr << file << ":" << line
			<< " CUDA call failed with error: "
			<< hipGetErrorString(err)
			<< std::endl;
		std::terminate();
	}
}

#define CHECK(x) check(__FILE__, __LINE__, (x))

__global__ void sum(const float *a, float *b) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int n = blockDim.x * gridDim.x;
	float sum = 0;
	int idx = col;
	for (int row = 0; row < n; row++) {
		sum += a[idx];
		idx += n;
	}
	b[col] = sum;
}
int main() {
	const int N = 8192;
	float *ha, *hb;
	float *da, *db;

	ha = new float [N * N];
	hb = new float [N];
	CHECK(hipMalloc(&da, N * N * sizeof(float)));
	CHECK(hipMalloc(&db, N * sizeof(float)));

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			ha[i * N + j] = sin(i * N + j);

	CHECK(hipMemcpy(da, ha, N * N * sizeof(float), hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	CHECK(hipEventRecord(start, 0));

	dim3 block(1024);
	dim3 grid(N / block.x);
	sum<<<grid, block>>>(da, db);

	CHECK(hipEventRecord(stop, 0));

	CHECK(hipEventSynchronize(stop));

	float timems;
	CHECK(hipEventElapsedTime(&timems, start, stop));
	std::cout << "Kernel elapsed time: " << timems << " ms" << std::endl;

	CHECK(hipMemcpy(hb, db, N * sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < 10; i++)
		std::cout << "b[" << i << "] = " << hb[i] << std::endl; 
	std::cout << "..." << std::endl;
	for (int i = N - 10; i < N; i++)
		std::cout << "b[" << i << "] = " << hb[i] << std::endl; 

	CHECK(hipFree(da));
	CHECK(hipFree(db));
	delete[] ha;
	delete[] hb;

	return 0;
}
