
#include <hip/hip_runtime.h>
#include <iostream>

void check(const char *file, const int line, hipError_t err) {
	if (err != hipSuccess) {
		std::cerr << file << ":" << line
			<< " CUDA call failed with error: "
			<< hipGetErrorString(err)
			<< std::endl;
		std::terminate();
	}
}

#define CHECK(x) check(__FILE__, __LINE__, (x))



__global__ void sum(const float *a, float *b) {


__shared__ float sa[384][32];
int row = threadIdx.x + blockIdx.x * blockDim.x;
int lane = threadIdx.x % 32; // номер нити в варпе
float sum = 0;
int N = blockDim.x * gridDim.x;
for (int col = 0; col < N; col += 32) {
	for (int j = 0; j < 32; j++)
	sa[threadIdx.x - lane + j][lane] = a[(row - lane + j) * N + col + lane];
	__syncthreads();
	for (int j = 0; j < 32; j++) sum += sa[threadIdx.x][j];
	__syncthreads();
	}
b[row] = sum;
}






int main() {
	const int N = 8192;
	float *ha, *hb;
	float *da, *db;

	ha = new float [N * N];
	hb = new float [N];
	CHECK(hipMalloc(&da, N * N * sizeof(float)));
	CHECK(hipMalloc(&db, N * sizeof(float)));

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			ha[i * N + j] = sin(i * N + j);

	CHECK(hipMemcpy(da, ha, N * N * sizeof(float), hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	CHECK(hipEventRecord(start, 0));

	dim3 block(256);
	dim3 grid(N / block.x);
	sum<<<grid, block>>>(da, db);

	CHECK(hipEventRecord(stop, 0));

	CHECK(hipEventSynchronize(stop));

	float timems;
	CHECK(hipEventElapsedTime(&timems, start, stop));
	std::cout << "Kernel elapsed time: " << timems << " ms" << std::endl;

	CHECK(hipMemcpy(hb, db, N * sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < 10; i++)
		std::cout << "b[" << i << "] = " << hb[i] << std::endl; 
	std::cout << "..." << std::endl;
	for (int i = N - 10; i < N; i++)
		std::cout << "b[" << i << "] = " << hb[i] << std::endl; 

	CHECK(hipFree(da));
	CHECK(hipFree(db));
	delete[] ha;
	delete[] hb;

	return 0;
}
